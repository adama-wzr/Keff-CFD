#include "keff2D.cuh"

int main(void){

	// Important call for efficiency on Linux
	fflush(stdout);

	//	Declare data structure
	options opts;

	char inputFilename[30];

	sprintf(inputFilename, "input.txt");

	readInputFile(inputFilename, &opts);

	int numDevices = 0;

	hipGetDeviceCount(&numDevices);
	if(opts.verbose == 1){
		printf("Cuda Capable Devices = %d\n", numDevices);
	}

	if(opts.BatchFlag == 0){
		SingleSim(opts);
	}else if(opts.BatchFlag == 1){
		BatchSim(opts);
	} else{
		std::cout << "Error: no valid BatchFlag option, check input file." << std::endl;
	}

	return 0;

}